#include "hip/hip_runtime.h"
/* 
 *  Jan Totz <jantotz@itp.tu-berlin.de>
 */

#include <vector>								// vector (for tip finder)
#include <iostream>								// cout, endl
#include <list>									// list (for fila finder)
#include <string>								// string (in struct of hpp)
#include <stdio.h>								// printf
#include <fstream>								// ofsream, ifstream

#include "SWC_GPU_solver.hpp"
#include "safe.hpp"
#include "vector_types_operator_overloads.hu"


// cuda error checking function
// usage: checkCUDAError("test",__LINE__);
void checkCUDAError(const char *msg, int line) {
#ifdef DEBUG
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if ( err != hipSuccess){
		fprintf(stderr, "Cuda error: line %d: %s: %s.\n", line, msg, hipGetErrorString( err) );
		exit(EXIT_FAILURE);
	}
#endif
}


// structure holds all device pointers
// simplifies pointer handling
struct device_pointers {
	
	// arrays
	Real *c0;
	Real *c;
	Real *cnew;
	Real *cdelay;
	Real *k;
	Real *output;
	Real *mask;
	
	// previously arrays, now a single value for faster access speed
	Real2 coupling_coeffs2;
	
};

struct host_pointers {
	
	Real *c;
	Real *ctemp;
	Real *mask;
	int *defects;
	
};

class streams {
	
	public:
	streams(){
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
	}
	
	~streams(){
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);
	}
	
	hipStream_t stream1;
	hipStream_t stream2;
	
};


// CPU pointer swap function for GPU
template <typename T>
void swapGPU(T &a, T &b){
	T t = a;
	a = b;
	b = t;
}



// fitz hugh nagumo, classic nerve conduction + heart + bz (zykov)
__global__ void model_fhn(Real2 *c, Real2 *cnew, int len, Real dt){
	Real ooeps=1.0/0.05;
	Real a=0.9;			// oscillatory
	
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i<len){
		cnew[i].x=c[i].x+dt*( ooeps*(c[i].x-1.0/3.0*c[i].x*c[i].x*c[i].x-c[i].y) );			// u
		cnew[i].y=c[i].y+dt*( c[i].x + a );													// v
	}
}



// zbke2k, more complete BZ model
// source: Taylor, Tinsley Toth
// must be double, not float!
__global__ void model_zbke2k(Real2 *c, Real2 *cnew, int len, Real dt){
	Real ooeps1=9.090909090909091;	// 1.0/0.11
	Real gammaEps2=2.04e-5;
	Real eps31=1.0016;
	Real alpha=0.1;
	Real beta=1.7e-5;
	
	Real mu=2.4e-4;
	Real q=0.7;
	Real phi=1.6e-4;
	
	Real uss=0.0;
	Real temp=0.0;

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i<len){
		uss=1.0/(4.0*gammaEps2) * (-(1.0-c[i].y) + sqrt(1.0 + fma(c[i].y,c[i].y,-2.0*c[i].y) + 16.0*gammaEps2*c[i].x));
		temp=alpha*c[i].y/(eps31-c[i].y);
		
		cnew[i].x=c[i].x+dt*( ooeps1*(phi-c[i].x*c[i].x-c[i].x+gammaEps2*uss*uss+uss*(1.0-c[i].y)+(mu-c[i].x)/(mu+c[i].x)*(q*temp+beta)) );			// x
		cnew[i].y=c[i].y+dt*(2.0*phi + uss*(1.0-c[i].y) - temp);																						// z
	}
}


// zbke2k, more complete BZ model
// source: Taylor Tinsley Toth paper
// must be double, not float!
__global__ void model_zbke2k_qhet(Real2 *c, Real2 *cnew, int len, Real dt, Real *het){
	Real ooeps1=9.090909090909091;	// 1.0/0.11
	Real gammaEps2=2.04e-5;
	Real eps31=1.0016;
	Real alpha=0.1;
	Real beta=1.7e-5;
	
	Real mu=2.4e-4;
	Real phi=1.6e-4;
	
	Real uss=0.0;
	Real temp=0.0;

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i<len){
		uss=1.0/(4.0*gammaEps2) * (-(1.0-c[i].y) + sqrt(1.0 + fma(c[i].y,c[i].y,-2.0*c[i].y) + 16.0*gammaEps2*c[i].x));
		temp=alpha*c[i].y/(eps31-c[i].y);
		
		cnew[i].x=c[i].x+dt*( ooeps1*(phi-c[i].x*c[i].x-c[i].x+gammaEps2*uss*uss+uss*(1.0-c[i].y)+(mu-c[i].x)/(mu+c[i].x)*(het[i]*temp+beta)) );			// x
		cnew[i].y=c[i].y+dt*(2.0*phi + uss*(1.0-c[i].y) - temp);																							// z
	}
}




void reaction(device_pointers *d, params &p, streams *s){
	
	
	int warpsize=32;
	dim3 nblocks((p.ncomponents*p.n-1)/warpsize+1);
	dim3 nthreads(warpsize);
	
	switch(p.reactionModel){
		case 24: model_zbke2k<<<nblocks,nthreads,0,s->stream1>>>((Real2 *)d->c,(Real2 *)d->cnew,p.n,p.dt); break;
		case 2401: model_zbke2k_qhet<<<nblocks,nthreads,0,s->stream1>>>((Real2 *)d->c,(Real2 *)d->cnew,p.n,p.dt,d->k); break;
		case 25: model_fhn<<<nblocks,nthreads,0,s->stream1>>>((Real2 *)d->c,(Real2 *)d->cnew,p.n,p.dt); break;
		default:
			printf("chosen reactionModel (%d) is not implemented! Program Abort!",p.reactionModel);
			exit(EXIT_FAILURE);
			break;
	}
	
	checkCUDAError("reaction()",__LINE__);
}




template <int BC>
__global__ void nonlocal_delay_homo_zbke2k_2d(Real2 *input, Real2 *output, Real2 *delay, int kdia, int kradius, Real ksum, int nx, int ny, const Real * __restrict__ M, const Real2 couplecoeff){
	
	// thread indices
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	
	if(x<nx && y<ny){
		int idx=x+y*nx;
		Real sum {};
		
		// global boundary conditions
		switch(BC){
			case 0:											// periodic
				for(int kx=0; kx<kdia; kx++){
				for(int ky=0; ky<kdia; ky++){
					int jx=x+kx-kradius;
					int jy=y+ky-kradius;
					if(jx<0){ jx+=nx; } else if(jx>=nx){ jx-=nx; }
					if(jy<0){ jy+=ny; } else if(jy>=ny){ jy-=ny; }
					sum += M[kx+ky*kdia]*delay[jx+jy*nx].y;
				}}
				sum -= ksum*input[idx].y;
				
				output[idx] += couplecoeff*sum;
				break;
			
			case 1:											// neumann
				for(int kx=0; kx<kdia; kx++){
				for(int ky=0; ky<kdia; ky++){
					int jx=x+kx-kradius;
					int jy=y+ky-kradius;
					if(jx >= 0 && jx < nx && jy >= 0 && jy < ny){
						sum += M[kx+ky*kdia]*(delay[jx+jy*nx].y - input[idx].y);
					}
				}}
				
				output[idx] += couplecoeff*sum;
				break;
		}
	}
}



template <int BC, int mask_radius>
__global__ void nonlocal_delay_homo_tiled_zbke2k_2d(Real2 *input, Real2 *output, Real2 *input_delay, int width, int height, 
int o_tile_width, const Real * __restrict__ M, const Real2 diffcoeff){
	
	// declare shared memory arrays for tiles, BLOCK_WIDTH = TILE_WIDTH, but != O_TILE_WIDTH
	extern __shared__ Real input_shared[];
	
	// thread indices, no dependence on blockIdx, blockDim to support tiling. 1 tile = 1 block
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	// row and column calculation, since in- and output tiles have different sizes
	int col_o = blockIdx.x*o_tile_width+tx;
	int row_o = blockIdx.y*o_tile_width+ty;
	int col_i = col_o-mask_radius;
	int row_i = row_o-mask_radius;
	
	// global boundary conditions
	int idx = col_i+row_i*width;
	
	Real output_temp{};
	switch(BC){
		case 0:									// periodic
			switch(mask_radius){
				case 5:
					if(col_i==-5) idx += width;
					if(col_i==width+4) idx -= width;
					if(row_i==-5) idx += width*height;
					if(row_i==height+4) idx -= width*height;
				case 4:
					if(col_i==-4) idx += width;
					if(col_i==width+3) idx -= width;
					if(row_i==-4) idx += width*height;
					if(row_i==height+3) idx -= width*height;
				case 3:
					if(col_i==-3) idx += width;
					if(col_i==width+2) idx -= width;
					if(row_i==-3) idx += width*height;
					if(row_i==height+2) idx -= width*height;
				case 2:
					if(col_i==-2) idx += width;
					if(col_i==width+1) idx -= width;
					if(row_i==-2) idx += width*height;
					if(row_i==height+1) idx -= width*height;
				case 1:
					if(col_i==-1) idx += width;
					if(col_i==width) idx -= width;
					if(row_i==-1) idx += width*height;
					if(row_i==height) idx -= width*height;
					break;
			}
			input_shared[tx+ty*blockDim.x] = input_delay[idx].y;
			__syncthreads();
			
			// calculation, not all threads are needed. Threads at tile boundaries are excluded.
			if(ty<o_tile_width && tx < o_tile_width){
				int mask_width=2*mask_radius+1;
				Real input0=input[row_o*width+col_o].y;
				for(int i=0; i<mask_width; i++){
				for(int j=0; j<mask_width; j++){
					output_temp += M[j*mask_width+i]*(input_shared[i+tx+blockDim.x*(j+ty)]-input0);
				}}
			}
			__syncthreads();
			
			break;
		
		case 1:																		// neumann
			if((row_i>=0) && (row_i<height) && (col_i>=0) && (col_i<width)){
				input_shared[tx+ty*blockDim.x] = input_delay[idx].y;
			}else{
				input_shared[tx+ty*blockDim.x] = 0.0;
			}
			__syncthreads();
			
			// calculation, not all threads are needed. Threads at tile boundaries are excluded.
			if(ty<o_tile_width && tx < o_tile_width){
				Real ksum{};
				int mask_width=2*mask_radius+1;
				Real input0=input[row_o*width+col_o].y;
				for(int i=0; i<mask_width; i++){
				for(int j=0; j<mask_width; j++){
					output_temp += M[j*mask_width+i]*input_shared[i+tx+blockDim.x*(j+ty)];
					ksum += M[j*mask_width+i]*!!input_shared[i+tx+blockDim.x*(j+ty)];
				}}
				output_temp -= ksum*input0;
			}
			__syncthreads();
			
			
			break;
	
	}
	
	// write output, exclude output from threads, which contributed to loading data into shared memory but did not calc output
	if(row_o<height && col_o<width && tx<o_tile_width && ty<o_tile_width){
		output[row_o*width+col_o] += diffcoeff*output_temp;
	}
}



void nonlocal_delay_homo_tiled_zbke2k(device_pointers *d, params &p, streams *s){
	
	
	int mem_size=0;
	
	int maskWidth=2*p.ChimeraCutOffRange+1;
	int o_TileWidth=p.blockWidth-maskWidth+1;
	
	dim3 nblocks((p.nx-1)/o_TileWidth+1);
	dim3 nthreads(p.blockWidth);
	
	if(p.bc=="periodic"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/o_TileWidth+1;
				nthreads.y=p.blockWidth;
				mem_size=p.blockWidth*p.blockWidth;
				switch(p.ChimeraCutOffRange){
					case 1: nonlocal_delay_homo_tiled_zbke2k_2d<0,1><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 2: nonlocal_delay_homo_tiled_zbke2k_2d<0,2><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 3: nonlocal_delay_homo_tiled_zbke2k_2d<0,3><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 4: nonlocal_delay_homo_tiled_zbke2k_2d<0,4><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 5: nonlocal_delay_homo_tiled_zbke2k_2d<0,5><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					default: printf("ChimeraCutOffRange must be < 6! Program Abort!"); exit(1); break;
				}
				break;
			default: printf("spaceDim is not chosen correctly for diffusion! Program Abort!"); exit(1); break;
		}
	}else if(p.bc=="neummann"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/o_TileWidth+1;
				nthreads.y=p.blockWidth;
				mem_size=p.blockWidth*p.blockWidth;
				switch(p.ChimeraCutOffRange){
					case 1: nonlocal_delay_homo_tiled_zbke2k_2d<1,1><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 2: nonlocal_delay_homo_tiled_zbke2k_2d<1,2><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 3: nonlocal_delay_homo_tiled_zbke2k_2d<1,3><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 4: nonlocal_delay_homo_tiled_zbke2k_2d<1,4><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 5: nonlocal_delay_homo_tiled_zbke2k_2d<1,5><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					default: printf("ChimeraCutOffRange must be < 6! Program Abort!"); exit(1); break;
				}
				break;
			default: printf("spaceDim is not chosen correctly for diffusion! Program Abort!"); exit(1); break;
		}
	}
	
	checkCUDAError("nonlocal_delay_homo_tiled_zbke2k()",__LINE__);
}


void nonlocal_delay_homo_zbke2k(device_pointers *d, params &p, streams *s){
	
	
	dim3 nblocks((p.nx-1)/p.blockWidth+1);
	dim3 nthreads(p.blockWidth);
	
	if(p.bc=="periodic"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/p.blockWidth+1;
				nthreads.y=p.blockWidth;
				nonlocal_delay_homo_zbke2k_2d<0><<<nblocks,nthreads>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.kdia,p.kradius,p.ksum,p.nx,p.ny,d->mask,d->coupling_coeffs2);
				break;
			default: printf("spaceDim is not chosen correctly for nonlocal_delay_homo_zbke2k! Program Abort!"); exit(1); break;
		}
	}else if(p.bc=="neumann"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/p.blockWidth+1;
				nthreads.y=p.blockWidth;
				nonlocal_delay_homo_zbke2k_2d<1><<<nblocks,nthreads>>>((Real2*)d->c,(Real2*)d->cnew,(Real2*)d->cdelay,p.kdia,p.kradius,p.ksum,p.nx,p.ny,d->mask,d->coupling_coeffs2);
				break;
			default: printf("spaceDim is not chosen correctly for nonlocal_delay_homo_zbke2k! Program Abort!"); exit(1); break;
		}
	}
	
	checkCUDAError("nonlocal_delay_homo_zbke2k()",__LINE__);
}





template <int BC>
__global__ void nonlocal_homo_zbke2k_2d(Real2 *input, Real2 *output, int kdia, int kradius, Real ksum, int nx, int ny, const Real * __restrict__ M, const Real2 couplecoeff){
	
	// thread indices
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	
	
	if(x<nx && y<ny){
		int idx=x+y*nx;
		Real sum {};
		
		// global boundary conditions
		switch(BC){
			case 0:											// periodic
				for(int kx=0; kx<kdia; kx++){
				for(int ky=0; ky<kdia; ky++){
					int jx=x+kx-kradius;
					int jy=y+ky-kradius;
					if(jx<0){ jx+=nx; } else if(jx>=nx){ jx-=nx; }
					if(jy<0){ jy+=ny; } else if(jy>=ny){ jy-=ny; }
					sum += M[kx+ky*kdia]*input[jx+jy*nx].y;
				}}
				sum -= ksum*input[idx].y;
				
				output[idx] += couplecoeff*sum;		// calculation
				break;
			
			case 1:											// neumann
				for(int kx=0; kx<kdia; kx++){
				for(int ky=0; ky<kdia; ky++){
					int jx=x+kx-kradius;
					int jy=y+ky-kradius;
					if(jx >= 0 && jx < nx && jy >= 0 && jy < ny){
						sum += M[kx+ky*kdia]*(input[jx+jy*nx].y - input[idx].y);
					}
				}}
				
				output[idx] += couplecoeff*sum;		// calculation
				break;
		}
	}
}



void nonlocal_homo_zbke2k(device_pointers *d, params &p, streams *s){
	
	
	dim3 nblocks((p.nx-1)/p.blockWidth+1);
	dim3 nthreads(p.blockWidth);
	
	if(p.bc=="periodic"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/p.blockWidth+1;
				nthreads.y=p.blockWidth;
				nonlocal_homo_zbke2k_2d<0><<<nblocks,nthreads>>>((Real2*)d->c,(Real2*)d->cnew,p.kdia,p.kradius,p.ksum,p.nx,p.ny,d->mask,d->coupling_coeffs2);
				break;
			default: printf("spaceDim is not chosen correctly for nonlocal_homo_zbke2k! Program Abort!"); exit(1); break;
		}
	}else if(p.bc=="neumann"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/p.blockWidth+1;
				nthreads.y=p.blockWidth;
				nonlocal_homo_zbke2k_2d<1><<<nblocks,nthreads>>>((Real2*)d->c,(Real2*)d->cnew,p.kdia,p.kradius,p.ksum,p.nx,p.ny,d->mask,d->coupling_coeffs2);
				break;
			default: printf("spaceDim is not chosen correctly for nonlocal_homo_zbke2k! Program Abort!"); exit(1); break;
		}
	}
	
	checkCUDAError("nonlocal_homo_zbke2k()",__LINE__);
}




template <int BC, int mask_radius>
__global__ void nonlocal_homo_tiled_zbke2k_2d(Real2 *input, Real2 *output, int width, int height,
 int o_tile_width, const Real* __restrict__ M, const Real2 diffcoeff){
	
	// declare shared memory arrays for tiles, BLOCK_WIDTH = TILE_WIDTH, but != O_TILE_WIDTH
	extern __shared__ Real input_shared[];
	
	// thread indices, no dependence on blockIdx, blockDim to support tiling. 1 tile = 1 block
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	// row and column calculation, since in- and output tiles have different sizes
	int col_o = blockIdx.x*o_tile_width+tx;
	int row_o = blockIdx.y*o_tile_width+ty;
	int col_i = col_o-mask_radius;
	int row_i = row_o-mask_radius;
	
	// global boundary conditions
	int idx = col_i+row_i*width;
	
	Real output_temp{};
	switch(BC){
		case 0:			// periodic
			switch(mask_radius){
				case 5:
					if(col_i==-5) idx += width;
					if(col_i==width+4) idx -= width;
					if(row_i==-5) idx += width*height;
					if(row_i==height+4) idx -= width*height;
				case 4:
					if(col_i==-4) idx += width;
					if(col_i==width+3) idx -= width;
					if(row_i==-4) idx += width*height;
					if(row_i==height+3) idx -= width*height;
				case 3:
					if(col_i==-3) idx += width;
					if(col_i==width+2) idx -= width;
					if(row_i==-3) idx += width*height;
					if(row_i==height+2) idx -= width*height;
				case 2:
					if(col_i==-2) idx += width;
					if(col_i==width+1) idx -= width;
					if(row_i==-2) idx += width*height;
					if(row_i==height+1) idx -= width*height;
				case 1:
					if(col_i==-1) idx += width;
					if(col_i==width) idx -= width;
					if(row_i==-1) idx += width*height;
					if(row_i==height) idx -= width*height;
					break;
			}
			input_shared[tx+ty*blockDim.x] = input[idx].y;
			__syncthreads();
			
			// calculation, not all threads are needed. Threads at tile boundaries are excluded.
			if(ty<o_tile_width && tx < o_tile_width){
				int mask_width=2*mask_radius+1;				// should be input value for speedup?
				Real input0=input_shared[tx+mask_radius + (ty+mask_radius)*blockDim.x];
				for(int i=0; i<mask_width; i++){
				for(int j=0; j<mask_width; j++){
					output_temp += M[j*mask_width+i]*(input_shared[i+tx+blockDim.x*(j+ty)]-input0);
				}}
			}
			__syncthreads();
			
			break;
		
		case 1:			// neumann
			if(col_i>=0 and col_i<width and row_i>=0 and row_i<height){
				input_shared[tx+ty*blockDim.x] = input[idx].y;
			}else{
				input_shared[tx+ty*blockDim.x] = 0.0;
			}
			__syncthreads();
			
			// calculation, not all threads are needed. Threads at tile boundaries are excluded.
			if(ty<o_tile_width && tx < o_tile_width){
				Real ksum{};
				int mask_width=2*mask_radius+1;				// should be input value for speedup?
				Real input0=input_shared[tx+mask_radius + (ty+mask_radius)*blockDim.x];
				for(int i=0; i<mask_width; i++){
				for(int j=0; j<mask_width; j++){
					output_temp += M[j*mask_width+i]*input_shared[i+tx+blockDim.x*(j+ty)];
					ksum += M[j*mask_width+i]*!!input_shared[i+tx+blockDim.x*(j+ty)];
				}}
				output_temp -= ksum*input0;
			}
			__syncthreads();
			
			break;
	}
	
	// write output, exclude output from threads, which contributed to loading data into shared memory but did not calc output
	if(row_o<height && col_o<width && tx<o_tile_width && ty<o_tile_width){
		output[row_o*width+col_o] += diffcoeff*output_temp;
	}
}




void nonlocal_homo_tiled_zbke2k(device_pointers *d, params &p, streams *s){
	
	
	int mem_size=0;
	
	int maskWidth=2*p.ChimeraCutOffRange+1;
	int o_TileWidth=p.blockWidth-maskWidth+1;
	
	
	dim3 nblocks((p.nx-1)/o_TileWidth+1);
	dim3 nthreads(p.blockWidth);
	
	if(p.bc=="periodic"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/o_TileWidth+1;
				nthreads.y=p.blockWidth;
				mem_size=p.blockWidth*p.blockWidth;	// number of elements
				switch(p.ChimeraCutOffRange){
					case 1: nonlocal_homo_tiled_zbke2k_2d<0,1><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 2: nonlocal_homo_tiled_zbke2k_2d<0,2><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 3: nonlocal_homo_tiled_zbke2k_2d<0,3><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 4: nonlocal_homo_tiled_zbke2k_2d<0,4><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 5: nonlocal_homo_tiled_zbke2k_2d<0,5><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					default: printf("ChimeraCutOffRange must be < 6! Program Abort!"); exit(1); break;
				}
				break;
			default:
				printf("spaceDim is not chosen correctly for diffusion! Program Abort!");
				exit(EXIT_FAILURE);
				break;
		}
	}else if(p.bc=="neumann"){
		switch(p.spaceDim){
			case 2:
				nblocks.y=(p.ny-1)/o_TileWidth+1;
				nthreads.y=p.blockWidth;
				mem_size=p.blockWidth*p.blockWidth;	// number of elements
				switch(p.ChimeraCutOffRange){
					case 1: nonlocal_homo_tiled_zbke2k_2d<1,1><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 2: nonlocal_homo_tiled_zbke2k_2d<1,2><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 3: nonlocal_homo_tiled_zbke2k_2d<1,3><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 4: nonlocal_homo_tiled_zbke2k_2d<1,4><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					case 5: nonlocal_homo_tiled_zbke2k_2d<1,5><<<nblocks,nthreads,mem_size*sizeof(Real),s->stream1>>>((Real2*)d->c,(Real2*)d->cnew,p.nx,p.ny,o_TileWidth,d->mask,d->coupling_coeffs2); break;
					default: printf("ChimeraCutOffRange must be < 6! Program Abort!"); exit(1); break;
				}
				break;
			default: printf("spaceDim is not chosen correctly for diffusion! Program Abort!"); exit(EXIT_FAILURE); break;
		}
	}
	
	checkCUDAError("nonlocal_homo_tiled_zbke2k()",__LINE__);
}











void coupling(device_pointers *d, params &p, streams *s, size_t step){
	
	// iterate over all components by changing the offset
	// move offset in array to address different components
	if(step>=p.stepsCouplingStart){
		switch(p.diffusionChoice){
			case 5: 											// nonlocal
				if(p.reactionModel==24 or p.reactionModel==2401 or p.reactionModel==25){
					if(p.use_tiles){ nonlocal_homo_tiled_zbke2k(d,p,s); }
					else{ nonlocal_homo_zbke2k(d,p,s); }
				}
				break;
			case 6:												// tnonlocal, delay; fill history until delaySteps+1: only local dynamics, no coupling
				if(p.delayHistoryUpdateStep>1) step = step/p.delayHistoryUpdateStep;
				
				if(step<p.delayStartSteps+1){
					if(p.reactionModel==24 or p.reactionModel==2401 or p.reactionModel==25){
						if(p.use_tiles){ nonlocal_homo_tiled_zbke2k(d,p,s); }
						else{ nonlocal_homo_zbke2k(d,p,s); }
					}
				}else if(step>=p.delayStartSteps+1){
					if(p.reactionModel==24 or p.reactionModel==2401 or p.reactionModel==25){
						if(p.use_tiles){ nonlocal_delay_homo_tiled_zbke2k(d,p,s); }
						else{ nonlocal_delay_homo_zbke2k(d,p,s); }
					}
				}
				break;
			case 7: break;										// no coupling
			default: printf("Error: diffusionChoice \"%d\" not implemented!\n",p.diffusionChoice); exit(1); break;
		}
	}
	
	checkCUDAError("diffusion()",__LINE__);
}


template <typename T>
__global__ void copyArrays(T *in, T *out, int len){
	
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i<len) out[i]=in[i];
}


Real kernelfunction(int i, int j, int i0, int j0, params &p){
	
	Real value=0.0;
	
	// exponential decay
	switch(p.spaceDim){
		case 2:
			value=p.dx*p.dy*exp(-sqrt((i-i0)*(i-i0)*p.dx*p.dx+(j-j0)*(j-j0)*p.dy*p.dy)/p.ChimeraKappa);
			break;
	}
	
	return value;
}

void create_kernel_and_rescale(int &n, params &p, host_pointers *h, modelparams &m){
	
	
	switch(p.diffusionChoice){
		
		// nonlocal BZ chimera coupling: Exp(-d/kappa), d = euclidean distance
		case 5:
		case 6:
			{
			int len=0, i=0, j=0, i0=0, j0=0;
			p.kdia=2*p.ChimeraCutOffRange+1;
			p.kradius = p.ChimeraCutOffRange;
			p.ksum = 0.0;
			switch(p.spaceDim){
				case 2:
					len=2*p.ChimeraCutOffRange+1;
					n=pow(len,2);
					i0=p.ChimeraCutOffRange; j0=i0;
					h->mask = new Real[n];
					for(i=0; i<len; i++){
					for(j=0; j<len; j++){
						h->mask[i+j*len] = kernelfunction(i,j,i0,j0,p);
					}}
					for(i=0; i<n; i++) h->mask[i] *= p.ChimeraK*p.dt;
					break;
			}
			// save kernel in binary data format for later
			std::ofstream dataout;
			dataout.open(p.pthout+"/coupling_kernel.bin",std::ios::binary);
			for(int i=0; i<n; i++) dataout.write((char*) &(h->mask[i]), sizeof(Real));
			dataout.close();
			for(int i=0; i<n; i++) p.ksum += h->mask[i];
			
			}
			break;
		
		// no rescaling
		case 7:					// no coupling
			break;
	}
}


void getArraySize(params &p, int &array_size){
	
	array_size=p.n*sizeof(Real2);
}






void cleanup_GPU(Real *c, device_pointers *d, params &p){
	
	printf("cleanup_GPU\n");
	hipError_t err;
	
	err = hipFree(d->k);
	err = hipFree(d->output);
	err = hipFree(d->mask);
	if(p.delayFlag==0){ err = hipFree(d->c); err = hipFree(d->cnew); }
	else if(p.delayFlag==1){ err = hipFree(d->c0); }
	
	// DEBUG
	if(err != hipSuccess){
		printf("Cuda error: %s\n",hipGetErrorString(err) );
		exit(EXIT_FAILURE);
	}
}


void copy_GPU_to_CPU(device_pointers *d, Real *c, params &p, streams *s){
	
	hipMemcpy(c,d->output,p.n*sizeof(Real2),hipMemcpyDeviceToHost);
	
}

// manage copy operation on GPU
void copy_GPU_to_GPU(device_pointers *d, params &p, streams *s){
	
	
	hipDeviceSynchronize();
	
	
	int warpsize=32;
	dim3 nblocks2((p.n-1)/warpsize+1,1,1);
	dim3 nthreads2(warpsize,1,1);
	
	copyArrays<<<nblocks2,nthreads2,0,s->stream1>>>((Real2*)d->c,(Real2*)d->output,p.n);
	
	checkCUDAError("copyArrays invocation",__LINE__);
	
	hipDeviceSynchronize();
}



void init_GPU(streams *s, params &p, Real *c, Real *k, device_pointers *d, modelparams &m){
	
	host_pointers h;
	h.c=c;
	int array_size=0;
	Real needed_mem=0.0;		// memory in bytes
	
	// datatype dependent improvements
	#ifdef DOUBLE
		hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);			// shared memory in 64 bit mode, better for double datatypes
	#endif
	if(p.use_tiles) hipDeviceSetCacheConfig(hipFuncCachePreferShared);				// get more shared memory, better for tiling
	
	// DEBUG: available memory
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem,&totalMem);
	printf("available/total GPU memory: %.2f/%.2f\n" ,freeMem/(1024.*1024.),totalMem/(1024.*1024.));
	
	// allocate GPU memory & move data from host to device
	getArraySize(p,array_size);
	if(p.delayFlag==0){												// no delay
		printf("c & cnew size: %.3f MB = states: 2\n",2*array_size/(1024.*1024.));
		needed_mem+=2*array_size;
		if(needed_mem/(1024.*1024.)>freeMem/(1024.*1024.)){ printf("Error: Too much GPU memory required! Abort now\n"); exit(1); }
		hipMalloc(&(d->c),array_size);
		hipMalloc(&(d->cnew),array_size);
		hipMemcpy(d->c,h.c,array_size,hipMemcpyHostToDevice);
	}else if(p.delayFlag==1){										// with delay
		size_t history_array_size=array_size*(p.delayStepsMax+1);
		printf("c0 size: %.3f MB = states: %zu\n",history_array_size/(1024.*1024.), p.delayStepsMax+1);
		needed_mem+=history_array_size;
		if(needed_mem/(1024.*1024.)>freeMem/(1024.*1024.)){ printf("Error: Too much GPU memory required! Abort now\n"); exit(1); }
		hipMalloc(&(d->c0),history_array_size);
		// init arrays for first step
		d->c = d->c0;
		hipMemcpy(d->c,h.c,array_size,hipMemcpyHostToDevice);
		d->cnew = d->c0+p.n*p.ncomponents;
	}else if(p.delayFlag==2){										// with delay, omitStates
		size_t history_array_size=array_size*(p.delayStepsMax+2);
		printf("c0 size: %.3f MB = states: %zu\n",history_array_size/(1024.*1024.), p.delayStepsMax+2);
		needed_mem+=history_array_size;
		if(needed_mem/(1024.*1024.)>freeMem/(1024.*1024.)){ printf("Error: Too much GPU memory required! Abort now\n"); exit(1); }
		hipMalloc(&(d->c0),history_array_size);
		// init arrays for first step
		d->c = d->c0+p.n*p.ncomponents;
		hipMemcpy(d->c,h.c,array_size,hipMemcpyHostToDevice);
		if(p.delayHistoryUpdateStep % 2 == 0){		// even
			d->cnew = d->c0;
		}else{										// uneven
			d->cnew = d->c0+(1+1)*p.n*p.ncomponents;
		}
	}
	printf("output size: %.3f MB\n",array_size/(1024.*1024.));
	needed_mem+=array_size;
	if(needed_mem/(1024.*1024.)>freeMem/(1024.*1024.)){ printf("Error: Too much GPU memory required! Abort now\n"); exit(1); }
	hipMalloc(&(d->output),array_size);
	
	
	// mask for convolutions and rescaling
	int maskSize=0;
	create_kernel_and_rescale(maskSize,p,&h,m);
	if( p.diffusionChoice==5 || p.diffusionChoice==6 ){
		printf("kernel size: %.3f MB\n",maskSize*sizeof(Real)/(1024.*1024.));
		needed_mem+=maskSize*sizeof(Real);
		if(needed_mem/(1024.*1024.)>freeMem/(1024.*1024.)){ printf("Error: Too much GPU memory required! Abort now\n"); exit(1); }
		hipMalloc(&(d->mask),maskSize*sizeof(Real));
		hipMemcpy(d->mask,h.mask,maskSize*sizeof(Real),hipMemcpyHostToDevice);
	}
	
	// set coupling coefficients' values
	d->coupling_coeffs2.x=m.coupling_coeffs[0]; 
	d->coupling_coeffs2.y=m.coupling_coeffs[1];
	
	
	
	int hetArraySize=p.n;
	printf("het-array size: %.3f MB\n",hetArraySize*sizeof(Real)/(1024.*1024.));
	needed_mem+=hetArraySize*sizeof(Real);
	if(needed_mem/(1024.*1024.)>freeMem/(1024.*1024.)){ printf("Error: Too much GPU memory required! Abort now\n"); exit(1); }
	hipMalloc(&(d->k),hetArraySize*sizeof(Real));
	hipMemcpy(d->k,k,hetArraySize*sizeof(Real),hipMemcpyHostToDevice);
	
	// info
	hipMemGetInfo(&freeMem,&totalMem);
	printf("available/total GPU memory: %.2f/%.2f\n" ,freeMem/(1024.*1024.),totalMem/(1024.*1024.));
	printf("total amount of GPU memory required: %.3f MB\n",needed_mem/(1024.*1024.));
	printf("total number of threads per block: %.0f <= 1024?\n",pow(p.blockWidth,p.spaceDim));
	if(pow(p.blockWidth,p.spaceDim)>1024){ printf("Error: Using too many threads per block!"); exit(EXIT_FAILURE);}
	
	checkCUDAError("init_GPU()",__LINE__);
	
}



void rd_dynamics(device_pointers *d, params &p, streams *s, size_t step){
	
	switch(p.delayFlag){
		case 0:											// no delay
			reaction(d,p,s);
			coupling(d,p,s,step);
			swapGPU(d->c,d->cnew);						// issue device pointer swap from host
			break;
		
		case 1:											// with delay
			reaction(d,p,s);
			coupling(d,p,s,step);
			// update pointer positions
			d->c = d->c0 + ((step+1) % (p.delayStepsMax+1))*p.n*p.ncomponents;
			d->cnew = d->c0 + ((step+2) % (p.delayStepsMax+1))*p.n*p.ncomponents;
			
			if(step>=p.delayStartSteps){
				size_t delaySteps=0;
				delaySteps=p.delayStepsMax;
				d->cdelay=d->c0+((step+3) % (delaySteps+1))*p.n*p.ncomponents;
			}
			
			break;
			
		case 2:											// with delay, omit steps for memory
			reaction(d,p,s);
			coupling(d,p,s,step);
			
			// pointer position iteration as update
			{
				int i=(step+1) % p.delayHistoryUpdateStep;
				size_t stepCoarse = (step+1) / p.delayHistoryUpdateStep;
				
				if(i==0){												// move data to next field (step 0)
					// array source: c
					d->c = d->c0 + p.n*p.ncomponents + (stepCoarse % (p.delayStepsMax+1))*p.n*p.ncomponents;
					// array target: cnew
					if(p.delayHistoryUpdateStep % 2 == 0){			// even
						d->cnew = d->c0;
					}else{											// odd
						d->cnew = d->c0 + p.n*p.ncomponents + ((stepCoarse+1) % (p.delayStepsMax+1))*p.n*p.ncomponents;
					}
					// move delay pointer further along
					if(stepCoarse>=p.delayStartSteps){
						size_t delaySteps=0;
						delaySteps=p.delayStepsMax;
						d->cdelay=d->c0+p.n*p.ncomponents + ((stepCoarse+2) % (delaySteps+1))*p.n*p.ncomponents;
					}
					
				}else if(i==1){											// set up start of swap cycle (step 1)
					d->c = d->cnew;
					// array target: cnew
					if(p.delayHistoryUpdateStep % 2 == 0){			// even
						d->cnew = d->c0 + p.n*p.ncomponents + ((stepCoarse+1) % (p.delayStepsMax+1))*p.n*p.ncomponents;
					}else{											// odd
						d->cnew = d->c0;
					}
				}else if(i>1){											// swap cycle (step 2 and more)
					swapGPU(d->c,d->cnew);
				}
			}
			break;
		
		default: printf("Unknown value for delayFlag (rd_dynamics).\n"); break;
	}
}




void solverGPU_2d(Real *c, Real *k, params &p, modelparams &m){
	
	printf("solverGPU_2d\n");
	
	// init
	device_pointers d;
	streams s;
	int untranslatedFlag=1;
	Real *ctemp = (Real *) calloc(p.n*p.ncomponents,sizeof(Real));
	
	// init class for saving
	int nSaveStates=100;
	Safe safe(p,nSaveStates);
	
	// save initial condition, ic=0
	translateArrayOrder(c,ctemp,p,untranslatedFlag);
	safe.save(ctemp,0);
	
	// prepare GPU
	init_GPU(&s,p,c,k,&d,m);
	
	// time loop
	for(size_t step=0; step<p.stepsEnd; step++){
		rd_dynamics(&d,p,&s,step);
		
		if(step>0 and (!(step%p.stepsSaveState))){
			
			// copy
			copy_GPU_to_GPU(&d,p,&s);
			copy_GPU_to_CPU(&d,c,p,&s);
			if(c[0]!=c[0]){ printf("step: %zu, u[0]=%f. Abort!\n",step,c[0]);  exit(EXIT_FAILURE); }
			
			// translate array from concentration major to space major
			translateArrayOrder(c,ctemp,p,untranslatedFlag);
			
			// save
			if(!(step%p.stepsSaveState)){ 
				if(untranslatedFlag){ safe.save(c,step); }
				else{ safe.save(ctemp,step); }
			}
		}
		
		// DEBUG
		checkCUDAError("Loop iteration",__LINE__);
	}
	
	// clean up data
	cleanup_GPU(c,&d,p);
	free(ctemp);
}



void solverGPU(Real *c, Real *k, params &p, modelparams &m){
	
	// serial version
	switch(p.spaceDim){
		case 2: solverGPU_2d(c,k,p,m); break;	// cartesian 2d
	}

}
